#include "hip/hip_runtime.h"
#include "client_cuda.hpp"

__global__ void AVG(const double *from, double *to, uint64_t N, uint64_t M) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    //printf("N %ld, M %ld\n", N, M);
    for (long long ti = tid; ti < N / M; ti += step)
    { // assuming N=M*x
        double sum = 0;
        for (long long fi = ti * M; fi < (ti + 1) * M; ++fi)
            sum += from[fi];
        to[ti] = sum / M;
    }
}

void gpuProcess(double *vect, double *avg_vect, uint64_t local_blocks, uint64_t M) {
    double *cu_vect, *cu_avg_vect;

    // Allocate device memory
    hipMalloc((void**)&cu_vect, local_blocks * M * sizeof(double));
    hipMalloc((void**)&cu_avg_vect, local_blocks * sizeof(double));
    // Transfer data from host to device memory
    hipMemcpy(cu_vect, vect, sizeof(double) * local_blocks * M, hipMemcpyHostToDevice);
    // Executing kernel 
    int cuBlock_size = 1024;
    int grid_size = (local_blocks + cuBlock_size) / cuBlock_size;
    AVG<<<grid_size,cuBlock_size>>>(cu_vect, cu_avg_vect, local_blocks * M, M);
    hipDeviceSynchronize();
    // Transfer data back to host memory
    hipMemcpy(avg_vect, cu_avg_vect, sizeof(double) * local_blocks, hipMemcpyDeviceToHost);
    // Deallocate device memory
    hipFree(cu_vect);
    hipFree(cu_avg_vect);
}